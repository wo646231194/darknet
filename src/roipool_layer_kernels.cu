#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "roipool_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"
#include "blas.h"
}

__global__ void forward_roipool_layer_kernel(int n, int in_h, int in_w, int in_c, int out_h, int out_w, float *roi, float *input, float *output, int *indexes, int num)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % out_w;
    id /= out_w;
    int i = id % out_h;
    id /= out_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;
    int s = b/num;

    int pool_index = j + out_w*(i + out_h*(k + in_c*b));

    int roi_start_w = roi[s*5+1] * in_w;
    int roi_start_h = roi[s*5+2] * in_h;
    // int roi_end_w = roi[s*5+3] * in_w;
    int roi_end_h = roi[s*5+4] * in_h;
    int roi_end_w = roi_end_h;

    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
	int roi_width = max(roi_end_w - roi_start_w + 1, 1);

    float bin_size_h = 1.0 * roi_height / out_h;//min is 1
    float bin_size_w = 1.0 * roi_width / out_w;//min is 1

    int hstart = floor(i*bin_size_h);
    int wstart = floor(j*bin_size_w);
    int hend = ceil((i+1)*bin_size_h);
    int wend = ceil((j+1)*bin_size_w);

    hstart = min(max(hstart + roi_start_h, 0), in_h);
    wstart = min(max(wstart + roi_start_w, 0), in_w);
    hend = min(max(hend + roi_end_h, 0), in_h);
    wend = min(max(wend + roi_end_w, 0), in_w);

    int is_empty = (hend <= hstart) || (wend <= wstart);

    float maxval = is_empty ? 0 : -INFINITY;
    input = input + s * in_h * in_w * in_c;
    for(int ih = hstart; ih < hend; ++ih){
        for(int iw = wstart; iw < wend; ++iw){
            int in = ih * in_w + iw;
            if(input[in] > maxval){
                maxval = input[in]; 
            }
        }
    }
    output[pool_index] = maxval;
}

// __global__ void backward_roipool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
// {
    
// }

extern "C" void forward_roipool_layer_gpu(roipool_layer layer, network_state state)
{
    int h = layer.out_h;
    int w = layer.out_w;
    int c = layer.c;

    size_t n = h*w*c*layer.batch;

    LAYER_TYPE type = state.net.layers[layer.index].type;
    if(type != CONVOLUTIONAL) printf("ROI index is not correct\n");
    float *conv = state.net.layers[layer.index].output_gpu;

    forward_roipool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.h, layer.w, layer.c, layer.out_h, layer.out_w, state.input, conv, layer.output_gpu, layer.indexes_gpu, layer.n);
    check_error(hipPeekAtLastError());
}

extern "C" void backward_roipool_layer_gpu(roipool_layer layer, network_state state)
{
    axpy_ongpu(layer.batch*layer.inputs, 1, layer.delta_gpu, 1, state.delta, 1);
    // size_t n = layer.h*layer.w*layer.c*layer.batch;

    // backward_roipool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.h, layer.w, layer.c, layer.stride, layer.size, layer.pad, layer.delta_gpu, state.delta, layer.indexes_gpu);
    // check_error(hipPeekAtLastError());
}

